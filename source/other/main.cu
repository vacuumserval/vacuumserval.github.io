#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 64 // 矩阵的宽度
#define BATCH 1 // 批处理大小

// CUDA 错误检查宏
#define CUDA_CHECK_ERROR(err) \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err)); \
        exit(-1); \
    }

// CUFFT 错误检查宏
#define CUFFT_CHECK_ERROR(err) \
    if (err != HIPFFT_SUCCESS) { \
        fprintf(stderr, "CUFFT Error: %d\n", err); \
        exit(-1); \
    }

int main() {
    hipfftHandle plan;
    hipfftReal *data;
    hipfftComplex *data_freq;
    hipfftReal *data_ifft;
    size_t alloc_size, work_size;
    hipfftResult cufft_status;
    hipError_t cuda_status;

    // 分配内存
    alloc_size = sizeof(hipfftReal) * NX * NX;
    cuda_status = hipMalloc((void **)&data, alloc_size);
    CUDA_CHECK_ERROR(cuda_status);

    alloc_size = sizeof(hipfftComplex) * (NX / 2 + 1) * NX;
    cuda_status = hipMalloc((void **)&data_freq, alloc_size);
    CUDA_CHECK_ERROR(cuda_status);

    alloc_size = sizeof(hipfftReal) * NX * NX;
    cuda_status = hipMalloc((void **)&data_ifft, alloc_size);
    CUDA_CHECK_ERROR(cuda_status);

    // 创建CUFFT计划
    cufft_status = hipfftPlan2d(&plan, NX, NX, HIPFFT_R2C);
    CUFFT_CHECK_ERROR(cufft_status);

    // 初始化矩阵并复制到设备内存
    hipfftReal h_data[NX * NX];
    for (int i = 0; i < NX * NX; ++i) {
        h_data[i] = (hipfftReal)i;
    }
    cuda_status = hipMemcpy(data, h_data, alloc_size, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR(cuda_status);

    // 执行前10个数字的输出
    hipfftReal h_data_output[NX * NX];
    hipMemcpy(h_data_output, data, 10 * sizeof(hipfftReal), hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_data_output[i]);
    }
    printf("\n");

    // 执行傅立叶变换
    cufft_status = hipfftExecR2C(plan, data, data_freq);
    CUFFT_CHECK_ERROR(cufft_status);

    // 归一化
    float norm_factor = 1.0 / (NX * NX);
    cuda_status = hipMemset(data_ifft, 0, alloc_size);
    CUDA_CHECK_ERROR(cuda_status);
    cufft_status = hipfftExecC2R(plan, data_freq, data_ifft);
    CUFFT_CHECK_ERROR(cufft_status);

    // 逆变换后输出前10个数字
    hipMemcpy(h_data_output, data_ifft, 10 * sizeof(hipfftReal), hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_data_output[i] * norm_factor);
    }
    printf("\n");

    // 释放资源
    hipfftDestroy(plan);
    hipFree(data);
    hipFree(data_freq);
    hipFree(data_ifft);

    return 0;
}
