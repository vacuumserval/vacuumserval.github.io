
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>

// 定义一些常量
#define NX 256 // 定义变换的x维度大小
#define NY 256 // 定义变换的y维度大小
#define BATCH 1 // 批处理大小

// CUDA错误检查宏
#define CUDA_CHECK_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// cuFFT错误检查宏
#define CUFFT_CHECK_ERROR(ans) { cufftAssert((ans), __FILE__, __LINE__); }
inline void cufftAssert(hipfftResult code, const char *file, int line, bool abort=true) {
   if (code != HIPFFT_SUCCESS) {
      fprintf(stderr,"CUFFTassert: %d %s %d\n", code, file, line);
      if (abort) exit(code);
   }
}

int main() {
    hipfftHandle plan;
    hipfftComplex *data;
    hipfftComplex *data_out;
    hipfftComplex *data_inverted;

    // 分配内存
    CUDA_CHECK_ERROR(hipMalloc((void**)&data, sizeof(hipfftComplex) * NX * NY));
    CUDA_CHECK_ERROR(hipMalloc((void**)&data_out, sizeof(hipfftComplex) * NX * NY));
    CUDA_CHECK_ERROR(hipMalloc((void**)&data_inverted, sizeof(hipfftComplex) * NX * NY));

    // 创建一个2D FFT计划
    CUFFT_CHECK_ERROR(hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C));

    // 初始化输入数据（这里仅作为示例，实际应用中应填充有意义的数据）
    hipfftComplex init_data[NX * NY];
    for (int i = 0; i < NX * NY; i++) {
        init_data[i].x = i; // 实部
        init_data[i].y = 0; // 虚部
    }

    // 将数据复制到GPU
    CUDA_CHECK_ERROR(hipMemcpy(data, init_data, sizeof(hipfftComplex) * NX * NY, hipMemcpyHostToDevice));

    // 执行正向FFT
    CUFFT_CHECK_ERROR(hipfftExecC2C(plan, data, data_out, HIPFFT_FORWARD));

    // 执行逆向FFT
    CUFFT_CHECK_ERROR(hipfftExecC2C(plan, data_out, data_inverted, HIPFFT_BACKWARD));

    // 将逆变换结果复制回主机
    CUDA_CHECK_ERROR(hipMemcpy(init_data, data_inverted, sizeof(hipfftComplex) * NX * NY, hipMemcpyDeviceToHost));

    // 归一化逆变换结果
    for (int i = 0; i < NX * NY; i++) {
        init_data[i].x /= (NX * NY);
        init_data[i].y /= (NX * NY);
    }

    // 打印逆变换结果的前几个元素
    for (int i = 0; i < 10; i++) {
        printf("Element %d: %f + %fi\n", i, init_data[i].x, init_data[i].y);
    }

    // 释放内存和计划
    CUDA_CHECK_ERROR(hipFree(data));
    CUDA_CHECK_ERROR(hipFree(data_out));
    CUDA_CHECK_ERROR(hipFree(data_inverted));
    CUFFT_CHECK_ERROR(hipfftDestroy(plan));

    return 0;
}
